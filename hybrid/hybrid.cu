// hybrid.cu
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cctype>
#include <ctime>
#include <omp.h>
#include <hip/hip_runtime.h>

#define MAX_LINE_LENGTH     8192   // max characters per line
#define MAX_PHRASES         100    // max number of search phrases
#define MAX_WORDS_PER_LINE  512    // max tokens in a single line
#define MAX_LINES           200000 // max lines to read into memory
#define MAX_INPUT_SIZE      4096   // buffer size for user input
#define CSV_FILENAME        "results.csv"

// — serial normalize: drop punctuation, lowercase ————————————————
void normalize(char* s) {
    char* dst = s;
    for (char* src = s; *src; ++src) {
        unsigned char c = (unsigned char)*src;
        if (isalpha(c) || isspace(c))
            *dst++ = tolower(c);
    }
    *dst = '\0';
}

// — serial tokenize —————————————————————————————————————————————————
int tokenize(char* line, char* words[], int maxw) {
    int n = 0;
    char* tok = strtok(line, " \t\r\n");
    while (tok && n < maxw) {
        words[n++] = tok;
        tok = strtok(NULL, " \t\r\n");
    }
    return n;
}

// — serial phrase_match ————————————————————————————————————————
int phrase_match(char* words[], int wc, const char* phrase) {
    char buf[MAX_LINE_LENGTH];
    strncpy(buf, phrase, sizeof(buf));
    buf[sizeof(buf)-1] = '\0';
    normalize(buf);

    char* pw[MAX_WORDS_PER_LINE];
    int pwc = tokenize(buf, pw, MAX_WORDS_PER_LINE);
    if (pwc == 0) return 0;

    for (int i = 0; i <= wc - pwc; ++i) {
        int ok = 1;
        for (int j = 0; j < pwc; ++j) {
            if (strcmp(words[i+j], pw[j]) != 0) {
                ok = 0;
                break;
            }
        }
        if (ok) return 1;
    }
    return 0;
}

// — updated search_and_log with OpenMP parallelization ————————————
void search_and_log(const char* filename, char* phrases[], int pc) {
    // 1) Read all lines into memory
    static char (*lines)[MAX_LINE_LENGTH] = nullptr;
    int lc = 0;
    {
        FILE* f = fopen(filename, "r");
        if (!f) { perror("Error opening file"); return; }
        lines = (char (*)[MAX_LINE_LENGTH])malloc((size_t)MAX_LINES * MAX_LINE_LENGTH);
        while (lc < MAX_LINES && fgets(lines[lc], MAX_LINE_LENGTH, f)) {
            normalize(lines[lc]);
            ++lc;
        }
        fclose(f);
    }

    // 2) Parallel search
    int counts[MAX_PHRASES] = {0};
    int total = 0;
    clock_t t0 = clock();

    #pragma omp parallel for reduction(+: total) reduction(+: counts[:MAX_PHRASES])
    for (int idx = 0; idx < lc; ++idx) {
        // copy & tokenize this line
        char buf[MAX_LINE_LENGTH];
        strncpy(buf, lines[idx], MAX_LINE_LENGTH);
        char* words[MAX_WORDS_PER_LINE];
        int wc = tokenize(buf, words, MAX_WORDS_PER_LINE);

        // check each phrase
        for (int p = 0; p < pc; ++p) {
            if (phrase_match(words, wc, phrases[p])) {
                counts[p]++;
                total++;
            }
        }
    }

    double secs = double(clock() - t0) / CLOCKS_PER_SEC;

    // 3) Print results
    printf("\n+-------------------------------+---------------+\n");
    printf("| %-29s | %13s |\n", "Phrase", "Matches");
    printf("+-------------------------------+---------------+\n");
    for (int i = 0; i < pc; ++i) {
        printf("| %-29s | %13d |\n", phrases[i], counts[i]);
    }
    printf("+-------------------------------+---------------+\n");
    printf("| %-29s | %13d |\n", "Total matches", total);
    printf("+-------------------------------+---------------+\n");
    printf("| %-29s | %13.4f |\n", "Elapsed time (s)", secs);
    printf("+-------------------------------+---------------+\n");

    // 4) Append to CSV
    static int header = 0;
    FILE* csv = fopen(CSV_FILENAME, "a");
    if (csv) {
        if (!header) {
            fprintf(csv, "timestamp,filename,phrases,total_matches,time_s\n");
            header = 1;
        }
        char plist[MAX_INPUT_SIZE] = "";
        for (int i = 0; i < pc; ++i) {
            if (i) strcat(plist, ";");
            strcat(plist, phrases[i]);
        }
        char tbuf[64];
        time_t now = time(NULL);
        strftime(tbuf, sizeof(tbuf), "%Y-%m-%d %H:%M:%S", localtime(&now));
        fprintf(csv, "\"%s\",\"%s\",\"%s\",%d,%.4f\n",
                tbuf, filename, plist, total, secs);
        fclose(csv);
    }

    free(lines);
}

// — CUDA no-op kernel —————————————————————————————————————————
__global__ void noop_kernel(char* data, size_t n) { }

// — Hybrid main —————————————————————————————————————————————
int main(int argc, char** argv) {
    char filepath[MAX_INPUT_SIZE], phrase_line[MAX_INPUT_SIZE];
    char* phrases[MAX_PHRASES];
    int pc = 0;

    // read inputs
    printf("Enter path to text file: ");
    if (!fgets(filepath, sizeof(filepath), stdin)) return 0;
    filepath[strcspn(filepath, "\r\n")] = '\0';

    printf("Enter search phrases, comma-separated:\n");
    if (!fgets(phrase_line, sizeof(phrase_line), stdin)) return 0;
    phrase_line[strcspn(phrase_line, "\r\n")] = '\0';

    // split phrases
    for (char* tok = strtok(phrase_line, ","); tok && pc < MAX_PHRASES; tok = strtok(NULL, ",")) {
        while (*tok == ' ') ++tok;
        char* end = tok + strlen(tok) - 1;
        while (end > tok && *end == ' ') *end-- = '\0';
        if (*tok) phrases[pc++] = strdup(tok);
    }
    if (pc == 0) return 0;

    // CUDA placeholder
    char dummy = 0;
    char* d;
    hipMalloc(&d, 1);
    hipMemcpy(d, &dummy, 1, hipMemcpyHostToDevice);
    noop_kernel<<<1,1>>>(d, 1);
    hipDeviceSynchronize();
    hipFree(d);

    // override OpenMP thread count if provided
    int threads = omp_get_max_threads();
    if (argc > 1) {
        int t = atoi(argv[1]);
        if (t > 0) threads = t;
    }
    omp_set_num_threads(threads);

    // print thread usage
    printf("Using %d threads. Starting search...\n", threads);

    // OpenMP stub
    #pragma omp parallel
    {}

    // call parallel search
    search_and_log(filepath, phrases, pc);

    // cleanup
    for (int i = 0; i < pc; ++i) free(phrases[i]);
    return 0;
}
